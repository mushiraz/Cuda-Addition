﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* c) {
	//Creates a list of threads 
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	return;
}

int main() {
	int a[] = { 1,2,3 };
	int b[] = { 4,5,6 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	

	// Creating pointers into the GPU
	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	//Allocate Memory in the GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));
	
	//copy the vectors into the gpu
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
	
	//vectorAdd<<<GRID_SIZE, BLOCK_SIZE
	// grid_size= # of threads to be generated, in this case we dont want the process to create too many threads and cause issues
	//BLOCK_SIZE= # of vectors which in this case is 3 

	vectorAdd <<<1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	/* Created a for loop to be able to add the two vectors and display the result in C
	
	for (int i = 0; i < sizeof(c) / sizeof(int); i++) {
		c[i] = a[i] + b[i];
	}*/
	return;

}